#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define DEBUG

inline void checkCudaError(const char* filename, const int linenum)
{
#ifdef DEBUG
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    printf("File: %s, line: %d, CUDA error : %s\n", filename, linenum, hipGetErrorString(error));
    exit(-1);
  }
#endif			
}

__global__ void foo(int *ptr)
{
  *ptr = 7;
}

int main(void)
{
  foo<<<1,1>>>(0);
  
  checkCudaError(__FILE__, __LINE__);
  
  return 0;
}
