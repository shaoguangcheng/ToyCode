#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define DEBUG

__global__ void add(const int* x, const int* y, int* z, const int n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < n)
    z[index] = x[index] + y[index];
}

void checkCudaError(const char* filename, const int linenum)
{
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    printf("File: %s, line: %d, CUDA error: %s\n", __FILE__, __LINE__, hipGetErrorString(error));
    exit(-1);
  }
}

void addVec(const int* x, const int*y, int* z, const int N)
{
  const int THREAD_PER_BLOCK = 512;
  const int nByte = N * sizeof(int);

  int *dev_x, *dev_y, *dev_z;

  hipMalloc((void**)(&dev_x), nByte);
  hipMalloc((void**)(&dev_y), nByte);
  hipMalloc((void**)(&dev_z), nByte);

  hipMemcpy(dev_x, x, nByte, hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, nByte, hipMemcpyHostToDevice);
  
  add<<<(N + THREAD_PER_BLOCK - 1)/THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(dev_x, dev_y, dev_z, N);

  hipMemcpy(z, dev_z, nByte, hipMemcpyDeviceToHost);

  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_z);
}

void random_int(int* array, int N)
{
  if(array){
    for(int i = 0; i < N; ++i)
      array[i] = rand()/1000;
  }
}

int main(void)
{
  const int N = 512*512;
  const int nByte = sizeof(int) * N;
  
  int *x = (int*)malloc(nByte);
  int *y = (int*)malloc(nByte);
  int *z = (int*)malloc(nByte);
  
  random_int(x, N);
  random_int(y, N);

  addVec(x, y, z, N);

  free(x);
  free(y);
  free(z);

  return 0;
}
